#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "matmul_cuda.h"
}


#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void matmul_gpu(float *x, float *w, float *output, unsigned int dim_1, unsigned int dim_2, unsigned int dim_3) {
    // x - dim1 x dim2
    // w - dim2 x dim3
    // output - dim1 x dim3 = xA
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < dim_1 && j < dim_3) {
        for (unsigned int k = 0; k < dim_2; k++) {
            output[i * dim_3 + j] += x[i * dim_2 + k] * w[k * dim_3 + j];
        }
    }
}



extern "C"
void matmul_gpu_flat(float *w, float *x, float *output, size_t dim_1, size_t dim_2, size_t dim_3) {
    // x - dim1 x dim2
    // w - dim2 x dim3
    // output - dim1 x dim3 = xA
    float *w_device, *x_device, *output_device;
    gpuErrchk(hipMalloc(&w_device, dim_2 * dim_3 * sizeof(float)));
    gpuErrchk(hipMalloc(&x_device, dim_1 * dim_2 * sizeof(float)));
    gpuErrchk(hipMalloc(&output_device, dim_1 * dim_3 * sizeof(float)));

    gpuErrchk(hipMemcpy(w_device, w, dim_2 * dim_3 * sizeof(float), hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy(x_device, x, dim_1 * dim_2 * sizeof(float), hipMemcpyHostToDevice))
    dim3 threadDim(32, 32);
    dim3 blockDim(dim_1 / 32 + 1, dim_3 / 32 + 1);
    matmul_gpu<<<blockDim, threadDim>>>(x_device, w_device, output_device, dim_1, dim_2, dim_3);
    gpuErrchk(hipMemcpy(output, output_device, dim_1 * dim_3 * sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(w_device));
    gpuErrchk(hipFree(x_device));
    gpuErrchk(hipFree(output_device));
}
